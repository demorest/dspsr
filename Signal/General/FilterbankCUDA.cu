#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2010 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

//#define _DEBUG 1

#include "dsp/FilterbankCUDA.h"
#include "CUFFTError.h"
#include "debug.h"

#include <hip/hip_runtime.h>

#include <iostream>
#include <assert.h>

void check_error_stream (const char*, hipStream_t);

#ifdef _DEBUG
#define CHECK_ERROR(x,y) check_error_stream(x,y)
#else
#define CHECK_ERROR(x,y)
#endif


__global__ void k_multiply (float2* d_fft, float2* kernel)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  float x = d_fft[i].x * kernel[i].x - d_fft[i].y * kernel[i].y;
  d_fft[i].y = d_fft[i].x * kernel[i].y + d_fft[i].y * kernel[i].x;
  d_fft[i].x = x;
}

__global__ void k_ncopy (float2* output_data, unsigned output_stride,
           const float2* input_data, unsigned input_stride,
           unsigned to_copy)
{
  output_data += blockIdx.y * output_stride;
  input_data += blockIdx.y * input_stride;

  unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < to_copy)
    output_data[index] = input_data[index];
}


using namespace std;

CUDA::FilterbankEngine::FilterbankEngine (hipStream_t _stream)
{
  real_to_complex = false;

  d_fft = d_kernel = 0;

  stream = _stream;

  nfilt_pos = 0;
  plan_fwd = 0;
  plan_bwd = 0;
  verbose = false;
}

CUDA::FilterbankEngine::~FilterbankEngine ()
{
}

void CUDA::FilterbankEngine::setup (dsp::Filterbank* filterbank)
{
  // A reference to the location of the dedispersion kernel on the GPU is
  // kept separate so that it only has to be loaded once
  float2** d_kernel_ptr = reinterpret_cast<float2**>(filterbank->get_d_kernel_gpu_ptr());
  d_kernel = *d_kernel_ptr;
  
  freq_res = filterbank->get_freq_res ();
  nchan_subband = filterbank->get_nchan_subband();

  real_to_complex = (filterbank->get_input()->get_state() == Signal::Nyquist);

  DEBUG("CUDA::FilterbankEngine::setup nchan_subband=" << nchan_subband \
        << " freq_res=" << freq_res);

  DEBUG("CUDA::FilterbankEngine::setup scratch=" << scratch);

  hipfftResult result;
  if (real_to_complex)
  {
    DEBUG("CUDA::FilterbankEngine::setup plan size=" << freq_res*nchan_subband*2);
    result = hipfftPlan1d (&plan_fwd, freq_res*nchan_subband*2, HIPFFT_R2C, 1);
    if (result != HIPFFT_SUCCESS)
      throw CUFFTError (result, "CUDA::FilterbankEngine::setup",
			"hipfftPlan1d(plan_fwd, HIPFFT_R2C)");
  }
  else
  {
    DEBUG("CUDA::FilterbankEngine::setup plan size=" << freq_res*nchan_subband);
    result = hipfftPlan1d (&plan_fwd, freq_res*nchan_subband, HIPFFT_C2C, 1);
    if (result != HIPFFT_SUCCESS)
      throw CUFFTError (result, "CUDA::FilterbankEngine::setup",
			"hipfftPlan1d(plan_fwd, HIPFFT_C2C)");
  }

  result = cufftSetCompatibilityMode(plan_fwd, CUFFT_COMPATIBILITY_FFTW_PADDING);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::FilterbankEngine::setup",
		      "cufftSetCompatibilityMode(plan_fwd)");

  DEBUG("CUDA::FilterbankEngine::setup setting stream=" << stream);
  result = hipfftSetStream (plan_fwd, stream);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::FilterbankEngine::setup", 
		      "hipfftSetStream(plan_fwd)");

  DEBUG("CUDA::FilterbankEngine::setup fwd FFT plan set");

  // if inverse FFT is necessary
  if (freq_res > 1)
  {
    int n[1] = { freq_res };
    result = hipfftPlanMany (&plan_bwd, 1, n, NULL, 0, 0, NULL, 0, 0,
                            HIPFFT_C2C, nchan_subband);
    if (result != HIPFFT_SUCCESS)
      throw CUFFTError (result, "CUDA::FilterbankEngine::setup",
            "hipfftPlanMany(plan_bwd)");

    // optimal performance for CUFFT regarding data layout
    result = cufftSetCompatibilityMode(plan_bwd, CUFFT_COMPATIBILITY_FFTW_PADDING);
    if (result != HIPFFT_SUCCESS)
      throw CUFFTError (result, "CUDA::FilterbankEngine::setup",
			"cufftSetCompatibilityMode(plan_bwd)");

    result = hipfftSetStream (plan_bwd, stream);
    if (result != HIPFFT_SUCCESS)
      throw CUFFTError (result, "CUDA::FilterbankEngine::setup",
			"hipfftSetStream(plan_bwd)");

    DEBUG("CUDA::FilterbankEngine::setup bwd FFT plan set");
  }

  nkeep = freq_res;

  multiply.init ();
  multiply.set_nelement(nchan_subband * freq_res);

  if (filterbank->has_response())
  {
    const dsp::Response* response = filterbank->get_response();
    
    if (!d_kernel)
    {
      unsigned nchan = response->get_nchan();
      unsigned ndat = response->get_ndat();
      unsigned ndim = response->get_ndim();

      assert( nchan == filterbank->get_nchan() );
      assert( ndat == freq_res );
      assert( ndim == 2 ); // complex

      unsigned mem_size = nchan * ndat * ndim * sizeof(hipfftReal);
        
      // allocate space for the convolution kernel
      hipMalloc (filterbank->get_d_kernel_gpu_ptr(), mem_size);
      d_kernel_ptr = reinterpret_cast<float2**>(filterbank->get_d_kernel_gpu_ptr());
      d_kernel = *d_kernel_ptr;
       
      // copy the kernel accross
      const float* kernel = filterbank->get_response()->get_datptr(0,0);
      
      if (stream)
        hipMemcpyAsync(d_kernel, kernel, mem_size, hipMemcpyHostToDevice, stream);
      else
        hipMemcpy (d_kernel, kernel, mem_size, hipMemcpyHostToDevice);  
    }
    
    nfilt_pos = response->get_impulse_pos();
    unsigned nfilt_tot = nfilt_pos + response->get_impulse_neg();

    // points kept from each small fft
    nkeep = freq_res - nfilt_tot;
  }

  if (!real_to_complex)
    return;
}

void CUDA::FilterbankEngine::set_scratch (float * _scratch)
{
  scratch = _scratch;
}

void CUDA::FilterbankEngine::finish ()
{
  check_error_stream ("CUDA::FilterbankEngine::finish", stream);
}


void CUDA::FilterbankEngine::perform (const dsp::TimeSeries * in, dsp::TimeSeries * out, 
            uint64_t npart, const uint64_t in_step, const uint64_t out_step)
{
  verbose = dsp::Operation::record_time || dsp::Operation::verbose;

  const unsigned npol = in->get_npol();
  const unsigned input_nchan = in->get_nchan();
  const unsigned output_nchan = out->get_nchan();
 
  // counters
  unsigned ipol, ichan;
  uint64_t ipart;
 
  // offsets into input and output
  uint64_t in_offset, out_offset;
  DEBUG("CUDA::FilterbankEngine::perform stream=" << stream);

  // GPU scratch space
  DEBUG("CUDA::FilterbankEngine::perform scratch=" << scratch);
  float2* cscratch = (float2*) scratch;

  hipfftResult result;
  float * output_ptr;
  float * input_ptr;
  uint64_t output_span;

  DEBUG("CUDA::FilterbankEngine::perform input_nchan=" << input_nchan);
  DEBUG("CUDA::FilterbankEngine::perform npol=" << npol);
  DEBUG("CUDA::FilterbankEngine::perform npart=" << npart);
  DEBUG("CUDA::FilterbankEngine::perform nkeep=" << nkeep);
  DEBUG("CUDA::FilterbankEngine::perform in_step=" << in_step);
  DEBUG("CUDA::FilterbankEngine::perform out_step=" << out_step);

  for (ichan=0; ichan<input_nchan; ichan++)
  {
    for (ipol=0; ipol < npol; ipol++)
    {
      for (ipart=0; ipart < npart; ipart++)
      {
        DEBUG("CUDA::FilterbankEngine::perform ipart " << ipart << " of " << npart);

        in_offset = ipart * in_step;
        out_offset = ipart * out_step;

        DEBUG("CUDA::FilterbankEngine::perform offsets in=" << in_offset << " out=" << out_offset);

        input_ptr = const_cast<float*>(in->get_datptr (ichan, ipol)) + in_offset;

        DEBUG("CUDA::FilterbankEngine::perform FORWARD FFT inptr=" << input_ptr << " outptr=" << cscratch);
        if (real_to_complex)
        {
          result = hipfftExecR2C(plan_fwd, input_ptr, cscratch);
          if (result != HIPFFT_SUCCESS)
            throw CUFFTError (result, "CUDA::FilterbankEngine::perform", "hipfftExecR2C");
          CHECK_ERROR ("CUDA::FilterbankEngine::perform hipfftExecR2C FORWARD", stream);
        }
        else
        {
          float2* cin = (float2*) input_ptr;
          result = hipfftExecC2C(plan_fwd, cin, cscratch, HIPFFT_FORWARD);
          if (result != HIPFFT_SUCCESS)
            throw CUFFTError (result, "CUDA::FilterbankEngine::perform", "hipfftExecC2C");
          CHECK_ERROR ("CUDA::FilterbankEngine::perform hipfftExecC2C FORWARD", stream);
        }

        if (d_kernel)
        {
          // complex numbers offset (d_kernel is float2*)
          unsigned offset = ichan * nchan_subband * freq_res; 
          DEBUG("CUDA::FilterbankEngine::perform multiply dedipersion kernel stream=" << stream);
          k_multiply<<<multiply.get_nblock(),multiply.get_nthread(),0,stream>>> (cscratch, d_kernel+offset);
          CHECK_ERROR ("CUDA::FilterbankEngine::perform multiply", stream);
        }

        if (plan_bwd)
        {
          DEBUG("CUDA::FilterbankEngine::perform BACKWARD FFT");
          result = hipfftExecC2C (plan_bwd, cscratch, cscratch, HIPFFT_BACKWARD);
          if (result != HIPFFT_SUCCESS)
            throw CUFFTError (result, "CUDA::FilterbankEngine::perform", "hipfftExecC2C (inverse)");

          CHECK_ERROR ("CUDA::FilterbankEngine::perform hipfftExecC2C BACKWARD", stream);
        }

        if (out)
        {
          output_ptr = out->get_datptr (ichan*nchan_subband, ipol) + out_offset;
          output_span = out->get_datptr (1, ipol) - out->get_datptr (0, ipol);

          const float2* input = cscratch + nfilt_pos;
          unsigned input_stride = freq_res;
          unsigned to_copy = nkeep;

          {
            dim3 threads;
            threads.x = multiply.get_nthread();

            dim3 blocks;
            blocks.x = nkeep / threads.x;
            if (nkeep % threads.x)
              blocks.x ++;

            blocks.y = nchan_subband;

            // divide by two for complex data
            float2* output_base = (float2*) output_ptr;
            unsigned output_stride = output_span / 2;

            DEBUG("CUDA::FilterbankEngine::perform output base=" << output_base << " stride=" << output_stride);
            DEBUG("CUDA::FilterbankEngine::perform input base=" << input << " stride=" << input_stride);
            DEBUG("CUDA::FilterbankEngine::perform to_copy=" << to_copy);

            k_ncopy<<<blocks,threads,0,stream>>> (output_base, output_stride,
                        input, input_stride, to_copy);
            CHECK_ERROR ("CUDA::FilterbankEngine::perform ncopy", stream);
          }
        } // if not benchmarking
      } // for each part
    } // for each polarization
  } // for each channel

  if (verbose)
    check_error_stream ("CUDA::FilterbankEngine::perform", stream);
}
